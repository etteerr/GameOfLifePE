#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "GoLgeneric.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <>

#define getl(X,Y) local[((X)+1) + (blockDim.x+2) * ((Y)+1)]

__global__ void cuda_kernel(int * src, int * dst, size_t width, size_t height) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
    dim3 li(threadIdx.x, threadIdx.y);

    extern __shared__ int local[];

    if (idx < width && idy < height) {
        getl(li.x, li.y) = get_rm(src, idx, idy);


        //Get edges
        
        /*
         0*****0    
         *     *
         *     *
         0*****0  
         */
        if (idx > 0 && li.x == 0)
            getl(li.x - 1, li.y) = get_rm(src, idx - 1, idy);

        if (idx < width - 1 && li.x == blockDim.x - 1)
            getl(li.x + 1, li.y) = get_rm(src, idx + 1, idy);

        if (idy > 0 && li.y == 0)
            getl(li.x, li.y - 1) = get_rm(src, idx, idy - 1);

        if (idy < height - 1 && li.y == blockDim.y - 1)
            getl(li.x, li.y + 1) = get_rm(src, idx, idy + 1);


        //Corners
        if ((li.x + li.y) == 0 && idx > 0 && idy > 0)
            getl(li.x - 1, li.y - 1) = get_rm(src, idx - 1, idy - 1);

        if (li.x == blockDim.x - 1 && li.y == blockDim.y - 1 && idx < width - 1 && idy < height - 1)
            getl(li.x + 1, li.y + 1) = get_rm(src, idx + 1, idy + 1);
    }
    __syncthreads();

    if (idx > 0 && idx < width - 1
            && idy > 0 && idy < height - 1) { //If we are not a edge

        int acc = 0;

        acc += getl(li.x - 1, li.y + 1);
        acc += getl(li.x - 1, li.y + 0);
        acc += getl(li.x - 1, li.y - 1);

        acc += getl(li.x - 0, li.y + 1);
        acc += getl(li.x - 0, li.y + 0);
        acc += getl(li.x - 0, li.y - 1);

        acc += getl(li.x + 1, li.y + 1);
        acc += getl(li.x + 1, li.y + 0);
        acc += getl(li.x + 1, li.y - 1);
        
        //acc = 2 : x * 1 + 0
        //acc = 3 : x * 0 + 1
        //acc = ? : x * 0 + 0
        get_rm(dst, idx, idy) = getl(li.x, li.y) * (int)(acc==2) + (int)(acc==3);
    }
}

__global__ void cuda_kernel_edge(int * src, int * dst, size_t width, size_t height) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; //Linear layout
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx == 0 || idy == 0 || idx == width - 1 || idy == height - 1) { //If we are on a edge
        if (idx < width && idy < height) {
            int acc = 0;

            size_t idxm1 = (size_t) (idx == 0) * width - 1 + (size_t) (idx > 0) * idx - 1;
            size_t idxp1 = (size_t) (idx + 1 < width) * (idx + 1);
            size_t idym1 = (size_t) (idy == 0) * height - 1 + (size_t) (idy > 0) * idy - 1;
            size_t idyp1 = (size_t) (idy + 1 < height) * (idy + 1);

            acc += get_rm(src, idxm1, idyp1);
            acc += get_rm(src, idxm1, idy + 0);
            acc += get_rm(src, idxm1, idym1);

            acc += get_rm(src, idx, idyp1);
            acc += get_rm(src, idx, idy + 0);
            acc += get_rm(src, idx, idym1);

            acc += get_rm(src, idxp1, idyp1);
            acc += get_rm(src, idxp1, idy + 0);
            acc += get_rm(src, idxp1, idym1);

            get_rm(dst, idx, idy) = get_rm(src, idx, idy) * (int)(acc==2) + (int)(acc==3);
        }
    }
}