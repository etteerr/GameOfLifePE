#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "GoLgeneric.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <>

__global__ void cuda_kernel(int * src, int * dst, size_t width, size_t height) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; //Linear layout
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx > 0 && idx < width - 1
            && idy > 0 && idy < height - 1) { //If we are not a edge
        int acc = 0;

        acc = get_rm(src, idx - 1, idy + 1);
        acc += get_rm(src, idx - 1, idy + 0);
        acc += get_rm(src, idx - 1, idy - 1);

        acc += get_rm(src, idx, idy + 1);
        acc += get_rm(src, idx, idy + 0);
        acc += get_rm(src, idx, idy - 1);

        acc += get_rm(src, idx + 1, idy + 1);
        acc += get_rm(src, idx + 1, idy + 0);
        acc += get_rm(src, idx + 1, idy - 1);

        if (acc == 2)
            get_rm(dst, idx, idy) = get_rm(src, idx, idy)!=0;
        else if (acc == 3)
            get_rm(dst, idx, idy) = 1;
        else
            get_rm(dst, idx, idy) = 0;
    }
}

__global__ void cuda_kernel_edge(int * src, int * dst, size_t width, size_t height) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; //Linear layout
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx == 0 || idy == 0 || idx == width - 1 || idy == height - 1) { //If we are on a edge
        if (idx < width && idy < height) {
            int acc = 0;
            if (idx > 0) {
                if (idy < height - 1) acc += get_rm(src, idx - 1, idy + 1);
                acc += get_rm(src, idx - 1, idy + 0);
                if (idy > 0) acc += get_rm(src, idx - 1, idy - 1);
            }
            if (idy < height - 1) acc += get_rm(src, idx, idy + 1);
            acc += get_rm(src, idx, idy + 0);
            if (idy > 0) acc += get_rm(src, idx, idy - 1);

            if (idx < width - 1) {
                if (idy < height - 1)acc += get_rm(src, idx + 1, idy + 1);
                acc += get_rm(src, idx + 1, idy + 0);
                if (idy > 0) acc += get_rm(src, idx + 1, idy - 1);
            }

            if (acc == 2)
                get_rm(dst, idx, idy) = get_rm(src, idx, idy)!=0;
            else if (acc == 3)
                get_rm(dst, idx, idy) = 1;
            else
                get_rm(dst, idx, idy) = 0;
        }
    }
}